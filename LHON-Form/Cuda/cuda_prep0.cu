
#include <hip/hip_runtime.h>
﻿
extern "C" __global__  void cuda_prep0(unsigned short im_size, int nerve_cent_pix, int nerve_r_pix_2, int vein_r_pix_2, float k_rate_extra, float k_detox_extra,
	unsigned char* pix_out_of_nerve, float* rate, float* detox)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < im_size && y < im_size) {

		int xy = x * im_size + y;
		int xy4 = xy * 4;

		int dx = x - nerve_cent_pix;
		int dy = y - nerve_cent_pix;
		int dis2 = dx * dx + dy * dy;

		bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
		pix_out_of_nerve[xy] = outside ? 1 : 0;

		if (!outside)
		{
			rate[xy4] = k_rate_extra;
			rate[xy4 + 1] = k_rate_extra;
			rate[xy4 + 2] = k_rate_extra;
			rate[xy4 + 3] = k_rate_extra;

			detox[xy] = k_detox_extra;
		}
	}
}

//bool[,] pix_out_of_nerve = new bool[im_size, im_size];
//for (int y = 0; y < im_size; y++)
//    for (int x = 0; x < im_size; x++)
//    {
//        int dx = x - nerve_cent_pix;
//        int dy = y - nerve_cent_pix;
//        int dis2 = dx * dx + dy * dy;

//        bool outside = nerve_r_pix_2 - dis2 < 0 || vein_r_pix_2 - dis2 > 0;
//        pix_out_of_nerve[x, y] = outside;
//        if (!outside)
//        {
//            pix_idx[pix_idx_num++] = x * im_size + y;
//            for (uint k = 0; k < 4; k++)
//                rate[x, y, k] = k_rate_extra;
//            detox[x, y] = k_detox_extra;
//        }
//    }

